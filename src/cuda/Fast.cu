#include "hip/hip_runtime.h"
#include "cuda/Fast.hpp"

#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

#include <opencv2/core/cuda/common.hpp>
#include <opencv2/core/cuda/utility.hpp>
#include <opencv2/core/cuda/reduce.hpp>
#include <opencv2/core/cuda/functional.hpp>

namespace ORB_SLAM2
{
namespace cuda
{
using namespace cv;
using namespace cv::cuda;

__global__
void detectKeypointsKernel(const unsigned int maxKeypoints,
                           const unsigned int highThres,
                           const unsigned int lowThres,
                           const PtrStepSz<unsigned char> image,
                           PtrStepSz<int> scoreMat,
                           KeyPoint *pKeypts)
{
    // set smem data types
    __shared__ int smem_warp_scan[32]; // for warp-level exclusive scan

    __shared__ int smem_block_count;

    cg::thread_block cta = cg::this_thread_block();

    const uint warp_id = cta.thread_rank() / warpSize;
    const uint block_warp_size = cta.size() / warpSize;

    const int j = threadIdx.x + blockIdx.x * blockDim.x + 3;
    const int i = (threadIdx.y + blockIdx.y * blockDim.y) * 4 + 3;

    bool isKeyPt[4] = {false, false, false, false};

    if (cta.thread_rank() == 0)
    {
        smem_block_count = 0;
    }

    cta.sync();

    for (int t = 0; t < 4; ++t)
    {
        if (i+t < image.rows-3 &&
            j < image.cols-3)
        {
            isKeyPt[t] = isKeyPoint2(image, i+t, j, highThres, scoreMat);
        }
    }

    cta.sync();

    for (int t = 0; t < 4; ++t)
    {
        short2 loc = make_short2(j, i+t);
        int score = scoreMat(loc.y, loc.x);

        bool findKeyPt = (isKeyPt[t] && isMax(loc));

        cg::coalesced_group active = cg::coalesced_threads();

        uint mask = active.ballot(findKeyPt);

        int total = __popc(mask);

        // if (active.thread_rank() == 0)
        //     smem_warp_scan[warp_id] = total;

        // cta.sync();

        // if (warp_id == 0)
        // {
        //     int warp_old = 0;

        //     // inclusive scan
        //     int warp_raw = (active.thread_rank() < block_warp_size) ? smem_warp_scan[active.thread_rank()] : 0;
        //     int warp_sum = warp_raw;

        //     uint valid_sum_mask = active.ballot(warp_sum > 0);

        //     if (__popc(valid_sum_mask) > 0)
        //     {
        //     for (int i = 1; i < block_warp_size; i*=2)
        //     {
        //         int prev = active.shfl_up(warp_sum, i);

        //         if (active.thread_rank() >= i) warp_sum += prev;
        //     }

        //     // atomic add based on shared memory
        //     if (active.thread_rank() == (block_warp_size-1))
        //     {
        //         warp_old = smem_block_count;

        //         smem_block_count += warp_sum;
        //     }

        //     // broadcast
        //     warp_old = active.shfl(warp_old, block_warp_size-1);
        //     }

        //     // convert to exclusive scan
        //     warp_sum -= warp_raw;

        //     smem_warp_scan[active.thread_rank()] = warp_sum + warp_old;
        // }

        isKeyPt[t] = false;
    }
}

GpuFast::GpuFast(int hThres, int lThres, int maxKeypoints)
    : mHighThres(hThres), mLowThres(lThres), mMaxKeypoints(maxKeypoints)
{
    checkCudaErrors(hipStreamCreate(&mCudaStream));
    mCvStream = StreamAccessor::wrapStream(mCudaStream);

    checkCudaErrors(hipMalloc(&mpKeyptsDevice, sizeof(KeyPoint) * mMaxKeypoints));
}

GpuFast::~GpuFast()
{
    mCvStream.~Stream();
    checkCudaErrors(hipFree(mpKeyptsDevice));
    checkCudaErrors(hipStreamDestroy(mCudaStream));
}

void GpuFast::detect(InputArray img,
                     std::vector<KeyPoint> &keypoints)
{
    this->detectAsync(img);

    this->joinDetectAsync(keypoints);
}

void GpuFast::detectAsync(InputArray img)
{
    const GpuMat image = img.getGpuMat();

    if (scoreMat.empty())
    {
        scoreMat = GpuMat(image.size(), CV_32SC1)
    }

    scoreMat.setTo(Scalar::all(0), mCvStream);

    dim3 dimBlock(32, 8);
    dim3 dimGrid(divUp(image.cols, dimBlock.x), divUp(image.rows, dimBlock.y * 4));

    // Use 7x7 kernel to detect FAST and Harris feature (the reason of the +3/-3 and *4/+4 tricks)
    detectKeypointsKernel<<<dimGrid, dimBlock, 0, mCudaStream>>>(
        mMaxKeypoints,
        mHighThres,
        mLowThres,
        image,
        scoreMat,
        mpKeyptsDevice
    );
    checkCudaErrors(hipGetLastError());
}

} // namespace cuda
} // namespace ORB_SLAM2